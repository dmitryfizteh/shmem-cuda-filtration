#include "hip/hip_runtime.h"
#include "defines.h"
#include <hip/hip_runtime.h>

__constant__ consts gpu_def [1];

// �������� ������ GPU
void checkErrors(char *label) 
{
#ifdef TEST
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
	}

	err=hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
		fflush(stdout);
	}
#endif
}

// �������� �� ����� �������� (�.�. �� ��������������� ������ ��� ������ �� ��������)
__device__ int device_is_active_point(int i, int localNx, int rank, int size)
{
	if((rank!=0 && i==0) || (rank!=size-1 && i==localNx-1))
		return 0;
	else
		return 1;
}

// �������������� ��������� ��������� ���������� � ����������
// ������ ��������� �������� �������������� ����� � ������� ���
// ������ �������, ���� ����� ������ 
// (���� 2 ������ � ����� ������,�� +2 �����). 
// ���������� ������� �������� ��� ������� ����� (������ � ������� �� rank==0)
__device__ int device_i_to_I(int i, int rank, int size, consts def)
{
	int I;
	if (rank <= ((*gpu_def).NX)%size)
	{
		if(rank==0)
			I=i;
		else
			I=(((*gpu_def).NX)/size+1)*rank+i-1;
	}
	else
		I=(((*gpu_def).NX)/size+1)*rank-(rank-((*gpu_def).NX)%size)+i-1;
	return I;
}

// ������ ����������, �������� NAPL P2 � Xi � ������ ����� ����� (���������� �� ��������� �����)
__global__ void assign_P2_Xi1_Xi2_ro1_ro2_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double S2 = DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P1 = DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S_e = (1 - S2 - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);
		double k1 = pow(S_e, (2. + 3. * (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]);
		double k2 = (1. - S_e) * (1. - S_e) * (1 - pow(S_e, (2. + (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]));
		double P_k = (*gpu_def).P_d[media] * pow((1 - S2 - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]), -1 / (*gpu_def).lambda[media]);

		DevArraysPtr.P2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P1 + P_k;
		DevArraysPtr.Xi1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k1 / mu1;
		DevArraysPtr.Xi2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k2 / mu2;
		DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = ro01 * (1 + ((*gpu_def).beta1) * (P1 - P_atm));
		DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = ro02 * (1 + ((*gpu_def).beta2) * (P1 + P_k - P_atm));
	}
}

// ������ ����������, �������� NAPL P2 � Xi �� ���� ������ �����
void ro_P2_Xi_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	assign_P2_Xi1_Xi2_ro1_ro2_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign P2, Xi,ro");
}

// ������ �������� � ������ ����� �����
__global__ void assign_u_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		double Xi1 = DevArraysPtr.Xi1[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double Xi2 = DevArraysPtr.Xi2[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P1 = DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P2 = DevArraysPtr.P2[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		if (((*gpu_def).NX)>2)
		{
			if (i == 0)
			{
				DevArraysPtr.u1x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * (DevArraysPtr.P1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P1) / ((*gpu_def).h1);
				DevArraysPtr.u2x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * (DevArraysPtr.P2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P2) / ((*gpu_def).h1);
			}
			if (i == localNx - 1)
			{
				DevArraysPtr.u1x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * (P1 - DevArraysPtr.P1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h1);
				DevArraysPtr.u2x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * (P2 - DevArraysPtr.P2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h1);
			}
			if ((i != 0) && (i != localNx - 1))
			{
				DevArraysPtr.u1x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * ((DevArraysPtr.P1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h1)));
				DevArraysPtr.u2x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * ((DevArraysPtr.P2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h1)));
			}
		}
		else
		{
			DevArraysPtr.u1x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.u2x[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}
	
		if (((*gpu_def).Ny)>2)
		{
			if (j == 0)
			{
				DevArraysPtr.u1y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * ((DevArraysPtr.P1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P1) / ((*gpu_def).h2) - DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.u2y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * ((DevArraysPtr.P2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P2) / ((*gpu_def).h2) - DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}

			if (j == ((*gpu_def).Ny) - 1)
			{
				DevArraysPtr.u1y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * ((P1 - DevArraysPtr.P1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h2) - DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.u2y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * ((P2 - DevArraysPtr.P2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h2) - DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}
			if ((j != 0) && (j != ((*gpu_def).Ny) - 1))
			{
				DevArraysPtr.u1y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * ((DevArraysPtr.P1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h2)) - DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.u2y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * ((DevArraysPtr.P2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h2)) - DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}
		}
		else
		{
			DevArraysPtr.u1y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.u2y[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

		if (((*gpu_def).Nz)>2)
		{
			if (k == 0)
			{
				DevArraysPtr.u1z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * (DevArraysPtr.P1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P1) / ((*gpu_def).h3);
				DevArraysPtr.u2z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * (DevArraysPtr.P2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P2) / ((*gpu_def).h3);
			}
			if (k == ((*gpu_def).Nz) - 1)
			{
				DevArraysPtr.u1z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * (P1 - DevArraysPtr.P1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h3);
				DevArraysPtr.u2z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * (P2 - DevArraysPtr.P2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).h3);
			}
			if ((k != 0) && (i != ((*gpu_def).Nz) - 1))
			{
				DevArraysPtr.u1z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi1 * ((DevArraysPtr.P1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h3)));
				DevArraysPtr.u2z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi2 * ((DevArraysPtr.P2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h3)));
			}
		}
		else
		{
			DevArraysPtr.u1z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.u2z[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}
	}
}

// ������ ��������� �� ���� ������ �����
void u_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	assign_u_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign u");
}
// ������ ro*S � ������ ����� ����� ������� ������������ ���������
__global__ void assign_rS_kernel_nr(ptr_Arrays DevArraysPtr, int localNx, double t)
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;
	
	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S2 = DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double roS1 = DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S2);
		double roS2 = DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S2;
 		double P1 = DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P2 = DevArraysPtr.P2[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double x1, x2, y1, y2, z1, z2, f1, f2, f3, g1, g2, g3;

		if (((*gpu_def).Nz)<2)
		{
			f3=0;
			g3=0;
		}
		else
		{
			z2 = -(DevArraysPtr.P1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).h3;
			z1 = -(P1 - DevArraysPtr.P1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).h3;

			f3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).h3;

			z2 = -(DevArraysPtr.P2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).h3;
			z1 = -(P2 - DevArraysPtr.P2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).h3;

			g3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).h3;
		}

		x2 = -(DevArraysPtr.P1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).h1;
        x1 = -(P1 - DevArraysPtr.P1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h1;

        y2 = -(DevArraysPtr.P1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).h2 + DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;
        y1 = -(P1 - DevArraysPtr.P1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h2 + DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;

        f1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h1;

        f2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h2;


        x2 = -(DevArraysPtr.P2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).h1;
        x1 = -(P2 - DevArraysPtr.P2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h1;

        y2 = -(DevArraysPtr.P2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).h2 + DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;
        y1 = -(P2 - DevArraysPtr.P2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h2 + DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;

        g1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h1;

        g2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).h2;

		DevArraysPtr.roS1_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1;
		DevArraysPtr.roS2_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2;
		DevArraysPtr.roS1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1 - ((*gpu_def).dt/(*gpu_def).m[media])*(f1 + f2 + f3);
		DevArraysPtr.roS2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2 - ((*gpu_def).dt/(*gpu_def).m[media])*(g1 + g2 + g3);
	}
}

// ������ ro*S � ������ ����� �����
__global__ void assign_rS_kernel(ptr_Arrays DevArraysPtr, int localNx, double t) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int local=i+j*localNx+k*localNx*((*gpu_def).Ny);
		int media = DevArraysPtr.media[local];
		double S2 = DevArraysPtr.S2[local];
		double roS1 = DevArraysPtr.ro1[local] * (1 - S2);
		double roS2 = DevArraysPtr.ro2[local] * S2;

		double divgrad1, divgrad2, Tx1, Ty1, Tx2, Ty2, Tz1, Tz2, A1=0, A2=0;

		if (((*gpu_def).Nz)<2)
		{
			divgrad1=0;
			divgrad2=0;
			Tz1=0;
			Tz2=0;
		}
		else
		{
			divgrad1 = ((*gpu_def).m[media] * ((*gpu_def).l_w) * ((*gpu_def).c) / 2.) * (DevArraysPtr.ro1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro1[local] * (1. - S2) + DevArraysPtr.ro1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).h3) * ((*gpu_def).h3));
			divgrad2 = ((*gpu_def).m[media] * ((*gpu_def).l_n) * ((*gpu_def).c) / 2.) * (DevArraysPtr.ro2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.S2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro2[local] * S2 + DevArraysPtr.ro2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (DevArraysPtr.S2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).h3) * ((*gpu_def).h3));
			Tz1 = (DevArraysPtr.ro1[i+1+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1x[i+1+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1x[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).h3));
			Tz2 = (DevArraysPtr.ro2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2y[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2y[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).h3));
		}

		divgrad1 += ((*gpu_def).m[media] * ((*gpu_def).l_w) * ((*gpu_def).c) / 2.) *
		((DevArraysPtr.ro1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S2) + DevArraysPtr.ro1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).h1) * ((*gpu_def).h1)) +
		(DevArraysPtr.ro1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro1[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S2) + DevArraysPtr.ro1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).h2) * ((*gpu_def).h2)));

		divgrad2 += ((*gpu_def).m[media] * ((*gpu_def).l_n) * ((*gpu_def).c) / 2.) *
		((DevArraysPtr.ro2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S2 + DevArraysPtr.ro2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).h1) * ((*gpu_def).h1)) +
		(DevArraysPtr.ro2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro2[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S2 + DevArraysPtr.ro2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).h2) * ((*gpu_def).h2)));

		Tx1 = (DevArraysPtr.ro1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1x[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1x[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h1));
		Ty1 = (DevArraysPtr.ro1[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1y[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u1y[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h2));
		Tx2 = (DevArraysPtr.ro2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2x[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2x[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h1));
		Ty2 = (DevArraysPtr.ro2[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2y[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.u2y[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).h2));

		if (t < 2 * ((*gpu_def).dt))
		{
			A1 = roS1 + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad1 - Tx1 - Ty1 - Tz1);
			A2 = roS2 + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad2 - Tx2 - Ty2 - Tz2);
		}
		else
		{
			A1 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad1 - Tx1 - Ty1 - Tz1 + (2 * roS1 * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS1_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
			A2 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad2 - Tx2 - Ty2 - Tz2 + (2 * roS2 * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS2_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
		}

		DevArraysPtr.roS1_old[local] = roS1;
		DevArraysPtr.roS2_old[local] = roS2;
		DevArraysPtr.roS1[local] = A1;
		DevArraysPtr.roS2[local] = A2;
	}
}

// ������ ro*S �� ���� ������ �����
void roS_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, double t, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	assign_rS_kernel_nr<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,t); 
	checkErrors("assign roS");
}

// ����� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void Newton_method_kernel(ptr_Arrays DevArraysPtr, int localNx) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
    {
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
        double S_e, P_k, AAA, F1, F2, PkS, F1P, F2P, F1S, F2S, det;
		double S2=DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P1=DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)];

        S_e = (1 - S2 - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);
        P_k = (*gpu_def).P_d[media] * pow(S_e, -1 / (*gpu_def).lambda[media]);
        AAA = pow(S_e, ((-1 / (*gpu_def).lambda[media]) - 1));
        F1 = ro01 * (1 + ((*gpu_def).beta1) * (P1 - P_atm)) * (1 - S2) - DevArraysPtr.roS1[i+j*localNx+k*localNx*((*gpu_def).Ny)];
        F2 = ro02 * (1 + ((*gpu_def).beta2) * (P1 + P_k - P_atm)) * S2 - DevArraysPtr.roS2[i+j*localNx+k*localNx*((*gpu_def).Ny)];

        PkS = AAA * (*gpu_def).P_d[media] / ((*gpu_def).lambda[media] * (1 - (*gpu_def).S_wr[media]));
        F1P = ro01 * ((*gpu_def).beta1) * (1 - S2);
        F2P = ro02 * ((*gpu_def).beta2) * S2;
        F1S = (-1) * ro01 * (1 + ((*gpu_def).beta1) * (P1 - P_atm));
        F2S = ro02 * (1 + ((*gpu_def).beta2) * (P1 + P_k - P_atm + (S2 * PkS)));

        det = F1P * F2S - F1S * F2P;

        DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P1 - (1 / det) * (F2S * F1 - F1S * F2);
        DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = S2 - (1 / det) * (F1P * F2 - F2P * F1);
    }
}

// ������ �������� ���� P1 � ������������ NAPL S2 �� ���� ������ �����
void P1_S2_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	Newton_method_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx); 
	checkErrors("assign P1 and S2");
}

// ��������� ������� �� S2
__global__ void S2_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && (((*gpu_def).NX)>2))
		{
		   DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S2[i+1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((i == localNx - 1) && (((*gpu_def).NX)>2))
		{
			DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S2[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S2[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			int I=device_i_to_I(i,rank,size, gpu_def[0]);
			if ((I>=((*gpu_def).NX)/2-((*gpu_def).source)) && (I<=((*gpu_def).NX)/2+((*gpu_def).source)) && (k>=((*gpu_def).Nz)/2-((*gpu_def).source)) && (k<=((*gpu_def).Nz)/2+((*gpu_def).source)))
				DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = S2_gr;
			else
				DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

		if ((k == 0) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S2[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)];
		}

		if ((k == ((*gpu_def).Nz) - 1) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.S2[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S2[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
		}
	}
}

// ��������� ������� �� P1
__global__ void P1_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && (((*gpu_def).NX)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P1[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]; 
		}

		if ((i == localNx - 1) && (((*gpu_def).NX)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P1[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P1[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] + DevArraysPtr.ro1[i+localNx*1] * g_const * ((*gpu_def).h2);
		}

		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_atm;
		}

		if ((k == 0) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P1[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]; 
		}

		if ((k == ((*gpu_def).Nz) - 1) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.P1[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P1[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
		}
	}
}

// ���������� ��������� �������
void boundary_conditions(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	S2_boundary_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign S2");
	P1_boundary_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign P1");
}	

// ������� �������� ������ � ������ �����
void load_data_to_host(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( HostArrayPtr, DevArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
}

// ������� �������� ������ ���� double � ������ ����������
void load_data_to_device(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy double data to device");
}

// ������� �������� ������ ���� int � ������ ����������
void load_data_to_device_int(int* HostArrayPtr, int* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(int), hipMemcpyHostToDevice );
	checkErrors("copy int data to device");
}

// ��������� ������ ���������� ��� ������ ����� ��������� �������
void device_memory_alloc(ptr_Arrays* ArraysPtr, double** DevBuffer, int localNx, consts def)
{
	hipMalloc((void**) DevBuffer,  2 * (def.Ny) * (def.Nz) * sizeof(double));

	hipMalloc((void**) &((*ArraysPtr).x),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).y),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).z),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P1),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P2),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).S2),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro1),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro2),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u1x),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u1y),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u1z),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u2x),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u2y),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).u2z),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi1),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi2),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS1),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS2),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS1_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS2_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).media),  localNx * (def.Ny) * (def.Nz) * sizeof(int));

	checkErrors("memory allocation");
}

// ����������� ������ ���������� �� ��� ������� ����� ��������� �������
void device_memory_free(ptr_Arrays DevArraysPtr, double* DevBuffer)
{
	hipFree (DevBuffer);
	hipFree (DevArraysPtr.x);
	hipFree (DevArraysPtr.y);
	hipFree (DevArraysPtr.z);
	hipFree (DevArraysPtr.P1);
	hipFree (DevArraysPtr.P2);
	hipFree (DevArraysPtr.S2);
	hipFree (DevArraysPtr.ro1);
	hipFree (DevArraysPtr.ro2);
	hipFree (DevArraysPtr.u1x);
	hipFree (DevArraysPtr.u1y);
	hipFree (DevArraysPtr.u1z);
	hipFree (DevArraysPtr.u2x);
	hipFree (DevArraysPtr.u2y);
	hipFree (DevArraysPtr.u2z);
	hipFree (DevArraysPtr.Xi1);
	hipFree (DevArraysPtr.Xi2);
	hipFree (DevArraysPtr.roS1);
	hipFree (DevArraysPtr.roS2);
	hipFree (DevArraysPtr.roS1_old);
	hipFree (DevArraysPtr.roS2_old);
	hipFree (DevArraysPtr.media);

	checkErrors("memory reliase");
}

// ������������� ����������
// ������ ���������� �� ����������, ����� �������� �����
// ������ ������������ ����������
void Device_Initialize(int rank, int* blocksX, int* blocksY, int* blocksZ, int localNx, consts def)
{
	// ���� 3 ���������� �� ����� ���� � ������� ����������� ����
	int device=rank%3;
	hipSetDevice(device);
	printf("CUDA initialized.\n");

	// ���������� ����������� ������
	// ���� ����� ����� ����� �� ������ ������� �����,
	// �� ���������� ������ ����� �� 1 ������.
	*blocksX=localNx/BlockNX;
	if (localNx%BlockNX!=0)
		(*blocksX)++;
	*blocksY=(def.Ny)/BlockNY;
	if ((def.Ny)%BlockNY!=0)
		(*blocksY)++;
	*blocksZ=(def.Nz)/BlockNZ;
	if ((def.Nz)%BlockNZ!=0)
		(*blocksZ)++;

	hipMemcpyToSymbol(HIP_SYMBOL( gpu_def), &def, sizeof ( consts ), 0, hipMemcpyHostToDevice );
	checkErrors("constant memory copy");

	int deviceCount;
    hipDeviceProp_t devProp;
    hipGetDeviceCount ( &deviceCount );

        hipGetDeviceProperties ( &devProp, device );
        printf ( "Device %d\n", device );
        printf ( "Compute capability : %d.%d\n", devProp.major, devProp.minor );
        printf ( "Name : %s\n", devProp.name );
        printf ( "Total Global Memory : %ld\n", devProp.totalGlobalMem );
        printf ( "Shared memory per block: %d\n", devProp.sharedMemPerBlock );
        printf ( "Registers per block : %d\n", devProp.regsPerBlock );
        printf ( "Warp size : %d\n", devProp.warpSize );
        printf ( "Max threads per block : %d\n", devProp.maxThreadsPerBlock );
        printf ( "Total constant memory : %d\n\n", devProp.totalConstMem );

		// ������������ ������ ��������� ����� ��� ����������
		// 21 - ���������� ���������� � �����
		printf ( "Total NAPL_Filtration grid size : %d\n\n", devProp.totalGlobalMem/(21*sizeof(double)) );

		// localNX+2 ������ ��� 2NyNz �� ����� ������ ����������
		if ((localNx+2)*(def.Ny)*(def.Nz) > (devProp.totalGlobalMem/(21*sizeof(double))))
			printf ("\nError! Not enough memory at GPU, rank=%d\n",rank);
}


__global__ void load_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		DevBuffer[j+((*gpu_def).Ny)*k]=DevArrayPtr[1+localNx*j+localNx*((*gpu_def).Ny)*k];
		DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)]=DevArrayPtr[localNx-2+localNx*j+localNx*((*gpu_def).Ny)*k];
	}
}


void load_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	load_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx); 
	checkErrors("load_exchange_data");
	hipMemcpy( HostBuffer, DevBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
}

__global__ void save_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx, int rank, int size)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		if (rank!=0)
			DevArrayPtr[localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k];
		if (rank!=size-1)
			DevArrayPtr[localNx-1+localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)];
	}
}

void save_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	hipMemcpy( DevBuffer, HostBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy data to device");
	save_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx, rank, size); 
	checkErrors("save_exchange_data");
}